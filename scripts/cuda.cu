#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel function: each thread squares one element
__global__ void square(float *d_out, float *d_in, int n) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < n) {
        d_out[idx] = d_in[idx] * d_in[idx];
    }
}

int main() {
    const int N = 16;
    size_t size = N * sizeof(float);

    // Host arrays
    float h_in[N], h_out[N];
    for (int i = 0; i < N; i++) h_in[i] = float(i);

    // Device arrays
    float *d_in, *d_out;
    hipMalloc(&d_in, size);
    hipMalloc(&d_out, size);

    // Copy data to device
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    // Launch kernel: <<<number of blocks, threads per block>>>
    int threadsPerBlock = 8;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    square<<<blocks, threadsPerBlock>>>(d_out, d_in, N);

    // Copy results back
    hipMemcpy(h_out, d_out, size, hipMemcpyDeviceToHost);

    // Print results
    for (int i = 0; i < N; i++) {
        std::cout << h_in[i] << "^2 = " << h_out[i] << std::endl;
    }

    // Free device memory
    hipFree(d_in);
    hipFree(d_out);
    return 0;
}
